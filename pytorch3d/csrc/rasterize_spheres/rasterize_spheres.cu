#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <math.h>
#include <cstdio>
#include <sstream>
#include <tuple>
#include "rasterize_points/bitmask.cuh"
#include "rasterize_spheres/rasterization_utils.cuh"

namespace {
// A little structure for holding details about a pixel.
struct Pix {
  float z; // Depth of the reference point.
  int32_t idx; // Index of the reference point.
  float dist2; // Euclidean distance square to the reference point.
};

__device__ inline bool operator<(const Pix& a, const Pix& b) {
  return a.z < b.z;
}

// This function checks if a pixel given by xy location pxy lies within the
// point with index p and batch index n. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the points which intersect
// with this pixel sorted by closest z distance. If the pixel pxy lies in the
// point, the list (q) is updated and re-orderered in place. In addition
// the auxillary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizeSpheresNaiveCudaKernel and
// RasterizeSpheresFineCudaKernel.
template <typename PointQ>
__device__ void CheckPixelInsidePoint(
    const float* points, // (P, 4)
    const int p_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    PointQ& q,
    const float xf, const float yf,
    const int K) {
  const float px = points[p_idx * 4 + 0];
  const float py = points[p_idx * 4 + 1];
  const float pz = points[p_idx * 4 + 2];
  const float pr = points[p_idx * 4 + 3];
  // Don't render points behind the camera or with no radius
  if (pz < 0 || pr <= 0) return;
  const float dx = xf - px;
  const float dy = yf - py;
  const float dist2 = dx * dx + dy * dy;
  if (dist2 < pr * pr) {
    if (q_size < K) {
      // Just insert it
      q[q_size] = {pz, p_idx, dist2};
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = q_size;
      }
      q_size++;
    } else if (pz < q_max_z) {
      // Overwrite the old max, and find the new max
      q[q_max_idx] = {pz, p_idx, dist2};
      q_max_z = pz;
      for (int i = 0; i < K; i++) {
        if (q[i].z > q_max_z) {
          q_max_z = q[i].z;
          q_max_idx = i;
        }
      }
    }
  }
}
} // namespace
// ****************************************************************************
// *                          NAIVE RASTERIZATION                             *
// ****************************************************************************

__global__ void RasterizeSpheresNaiveCudaKernel(
    const float* spheres, // (P, 4)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const int N, const int S, const int K,
    int32_t* point_idxs, // (N, S, S, K)
    float* zbuf, // (N, S, S, K)
    float* pix_dists) { // (N, S, S, K)
  // Simple version: One thread per output pixel
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = tid; i < N * S * S; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (S * S); // Batch index
    const int pix_idx = i % (S * S);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yi = S - 1 - pix_idx / S;
    const int xi = S - 1 - pix_idx % S;

    const float xf = PixToNdc(xi, S);
    const float yf = PixToNdc(yi, S);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    // TODO(jcjohns) Abstract this out into a standalone data structure
    Pix q[kMaxSpheresPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t point_start_idx = cloud_to_packed_first_idx[n];
    const int64_t point_stop_idx = point_start_idx + num_points_per_cloud[n];

    for (int p_idx = point_start_idx; p_idx < point_stop_idx; ++p_idx) {
      CheckPixelInsidePoint(spheres, p_idx, q_size, q_max_z, q_max_idx, q, xf, yf, K);
    }
    BubbleSort(q, q_size);
    int idx = n * S * S * K + pix_idx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizeSpheresNaiveCuda(
    const at::Tensor& points, // (P, 4)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const int image_size,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg
      points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizeSpheresNaiveCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 4,
      "points must have dimensions (num_points, 4)");
  TORCH_CHECK(
      num_points_per_cloud.size(0) == cloud_to_packed_first_idx.size(0),
      "num_points_per_cloud must have same size first dimension as cloud_to_packed_first_idx");

  const int N = num_points_per_cloud.size(0); // batch size.
  const int S = image_size;
  const int K = points_per_pixel;

  if (K > kMaxSpheresPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxSpheresPerPixel;
    AT_ERROR(ss.str());
  }

  auto int_opts = num_points_per_cloud.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, S, S, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, S, S, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, S, S, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizeSpheresNaiveCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      N, S, K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                          COARSE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizeSpheresCoarseCudaKernel(
    const float* spheres, // (P, 4)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const int N, const int P, const int S,
    const int bin_size,
    const int chunk_size,
    const int max_points_per_bin,
    int* points_per_bin,
    int* bin_points) {
  extern __shared__ char sbuf[];
  const int M = max_points_per_bin;
  const int num_bins = 1 + (S - 1) / bin_size; // Integer divide round up
  const float half_pix = 1.0f / S; // Size of half a pixel in NDC units

  // This is a boolean array of shape (num_bins, num_bins, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins, num_bins, chunk_size);

  // Have each block handle a chunk of points and build a 3D bitmask in
  // shared memory to mark which points hit which bins.  In this first phase,
  // each thread processes one point at a time. After processing the chunk,
  // one thread is assigned per bin, and the thread counts and writes the
  // points for the bin out to global memory.
  const int chunks_per_batch = 1 + (P - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;
  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch;
    const int chunk_idx = chunk % chunks_per_batch;
    const int point_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t cloud_point_start_idx = cloud_to_packed_first_idx[batch_idx];
    const int64_t cloud_point_stop_idx =
        cloud_point_start_idx + num_points_per_cloud[batch_idx];

    // Have each thread handle a different point within the chunk
    for (int p = threadIdx.x; p < chunk_size; p += blockDim.x) {
      const int p_idx = point_start_idx + p;

      // Check if point index corresponds to the cloud in the batch given by
      // batch_idx.
      if (p_idx >= cloud_point_stop_idx || p_idx < cloud_point_start_idx) {
        continue;
      }

      const float px = spheres[p_idx * 4 + 0];
      const float py = spheres[p_idx * 4 + 1];
      const float pz = spheres[p_idx * 4 + 2];
      const float radius = spheres[p_idx * 4 + 3];
      // Don't render points behind the camera or with radius 0
      if (pz < 0 || radius <= 0) continue;
      const float px0 = px - radius;
      const float px1 = px + radius;
      const float py0 = py - radius;
      const float py1 = py + radius;

      // Brute-force search over all bins; TODO something smarter?
      // For example we could compute the exact bin where the point falls,
      // then check neighboring bins. This way we wouldn't have to check
      // all bins (however then we might have more warp divergence?)
      for (int by = 0; by < num_bins; ++by) {
        // Get y extent for the bin. PixToNdc gives us the location of
        // the center of each pixel, so we need to add/subtract a half
        // pixel to get the true extent of the bin.
        const float by0 = PixToNdc(by * bin_size, S) - half_pix;
        const float by1 = PixToNdc((by + 1) * bin_size - 1, S) + half_pix;
        const bool y_overlap = (py0 <= by1) && (by0 <= py1);

        if (!y_overlap) continue;
        for (int bx = 0; bx < num_bins; ++bx) {
          // Get x extent for the bin; again we need to adjust the
          // output of PixToNdc by half a pixel.
          const float bx0 = PixToNdc(bx * bin_size, S) - half_pix;
          const float bx1 = PixToNdc((bx + 1) * bin_size - 1, S) + half_pix;
          const bool x_overlap = (px0 <= bx1) && (bx0 <= px1);

          if (x_overlap) binmask.set(by, bx, p);
        }
      }
    }
    __syncthreads();
    // Now we have processed every point in the current chunk. We need to
    // count the number of points in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins * num_bins; byx += blockDim.x) {
      const int by = byx / num_bins;
      const int bx = byx % num_bins;
      const int count = binmask.count(by, bx);
      const int points_per_bin_idx =
          batch_idx * num_bins * num_bins + by * num_bins + bx;

      // This atomically increments the (global) number of points found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_points array for the
      // points in the current chunk that fall into this bin.
      const int start = atomicAdd(points_per_bin + points_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_points.
      int next_idx = batch_idx * num_bins * num_bins * M + by * num_bins * M +
          bx * M + start;
      for (int p = 0; p < chunk_size; ++p) {
        if (binmask.get(by, bx, p)) {
          // TODO: Throw an error if next_idx >= M -- this means that
          // we got more than max_points_per_bin in this bin
          // TODO: check if atomicAdd is needed in line 265.
          bin_points[next_idx] = point_start_idx + p;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizeSpheresCoarseCuda(
    const at::Tensor& points, // (P, 4)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const int image_size,
    const int bin_size,
    const int max_points_per_bin) {
  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 4,
      "points must have dimensions (num_points, 4)");

  // Check inputs are on the same device
  at::TensorArg
      points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizeSpheresCoarseCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int P = points.size(0);
  const int N = num_points_per_cloud.size(0);
  const int num_bins = 1 + (image_size - 1) / bin_size; // divide round up
  const int M = max_points_per_bin;

  if (num_bins >= 22) {
    // Make sure we do not use too much shared memory.
    std::stringstream ss;
    ss << "Got " << num_bins << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_points_per_cloud.options().dtype(at::kInt);
  at::Tensor points_per_bin = at::zeros({N, num_bins, num_bins}, opts);
  at::Tensor bin_points = at::full({N, num_bins, num_bins, M}, -1, opts);

  if (bin_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_points;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins * num_bins * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizeSpheresCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      N, P,
      image_size,
      bin_size,
      chunk_size,
      M,
      points_per_bin.contiguous().data_ptr<int32_t>(),
      bin_points.contiguous().data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_points;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizeSpheresFineCudaKernel(
    const float* points, // (P, 4)
    const int32_t* bin_points, // (N, B, B, T)
    const int bin_size,
    const int N,
    const int B, // num_bins
    const int M, const int S, const int K,
    int32_t* point_idxs, // (N, S, S, K)
    float* zbuf, // (N, S, S, K)
    float* pix_dists) { // (N, S, S, K)
  // This can be more than S^2 if S is not dividable by bin_size.
  const int num_pixels = N * B * B * bin_size * bin_size;
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from points and bin_points.
    int i = pid;
    const int n = i / (B * B * bin_size * bin_size);
    i %= B * B * bin_size * bin_size;
    const int by = i / (B * bin_size * bin_size);
    i %= B * bin_size * bin_size;
    const int bx = i / (bin_size * bin_size);
    i %= bin_size * bin_size;

    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= S || xi >= S)
      continue;

    const float xf = PixToNdc(xi, S);
    const float yf = PixToNdc(yi, S);

    // This part looks like the naive rasterization kernel, except we use
    // bin_points to only look at a subset of points already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pix q[kMaxSpheresPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; ++m) {
      const int p = bin_points[n * B * B * M + by * B * M + bx * M + m];
      // bin_points uses -1 as a sentinal value
      if (p < 0) continue;
      CheckPixelInsidePoint(points, p, q_size, q_max_z, q_max_idx, q, xf, yf, K);
    }
    // Now we've looked at all the points for this bin, so we can write
    // output for the current pixel.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yidx = S - 1 - yi;
    const int xidx = S - 1 - xi;

    const int pix_idx = n * S * S * K + yidx * S * K + xidx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizeSpheresFineCuda(
    const at::Tensor& points, // (P, 4)
    const at::Tensor& bin_points,
    const int image_size,
    const int bin_size,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      bin_points_t{bin_points, "bin_points", 2};
  at::CheckedFrom c = "RasterizeSpheresFineCuda";
  at::checkAllSameGPU(c, {points_t, bin_points_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = bin_points.size(0);
  const int B = bin_points.size(1); // num_bins
  const int M = bin_points.size(3);
  const int S = image_size;
  const int K = points_per_pixel;
  if (K > kMaxSpheresPerPixel) AT_ERROR("Must have num_closest <= 150");
  auto int_opts = bin_points.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, S, S, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, S, S, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, S, S, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizeSpheresFineCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      bin_points.contiguous().data_ptr<int32_t>(),
      bin_size,
      N, B, M, S, K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO(T55115174) Add more documentation for backward kernel.
__global__ void RasterizeSpheresBackwardCudaKernel(
    const float* points, // (P, 4)
    const int32_t* idxs, // (N, H, W, K)
    const int N, const int P, const int H, const int W, const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_dists, // (N, H, W, K)
    float* grad_points) { // (P, 4)
  // Parallelized over each of K points per pixel, for each pixel in images of
  // size H * W, for each image in the batch of size N.
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < N * H * W * K; i += num_threads) {
    // const int n = i / (H * W * K); // batch index (not needed).
    const int yxk = i % (H * W * K);
    const int yi = yxk / (W * K);
    const int xk = yxk % (W * K);
    const int xi = xk / K;
    // k = xk % K (We don't actually need k, but this would be it.)
    // Reverse ordering of X and Y axes.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const float xf = PixToNdc(xidx, W);
    const float yf = PixToNdc(yidx, H);

    const int p = idxs[i];
    if (p < 0) continue;

    const float grad_dist2 = grad_dists[i];
    const int p_ind = p * 4; // index into packed points tensor
    const float px = points[p_ind + 0];
    const float py = points[p_ind + 1];
    const float dx = px - xf;
    const float dy = py - yf;
    const float grad_px = 2.0f * grad_dist2 * dx;
    const float grad_py = 2.0f * grad_dist2 * dy;
    const float grad_pz = grad_zbuf[i];

    atomicAdd(grad_points + p_ind + 0, grad_px);
    atomicAdd(grad_points + p_ind + 1, grad_py);
    atomicAdd(grad_points + p_ind + 2, grad_pz);
    // TODO add grad to radius here no idea what it is though
    // It's the last argument for future reference
    atomicAdd(grad_points + p_ind + 3, grad_px * grad_py);
  }
}

at::Tensor RasterizeSpheresBackwardCuda(
    const at::Tensor& points, // (N, P, 4)
    const at::Tensor& idxs, // (N, H, W, K)
    const at::Tensor& grad_zbuf, // (N, H, W, K)
    const at::Tensor& grad_dists) { // (N, H, W, K)

  // Check inputs are on the same device
  at::TensorArg
      points_t{points, "points", 1},
      idxs_t{idxs, "idxs", 2},
      grad_zbuf_t{grad_zbuf, "grad_zbuf", 3},
      grad_dists_t{grad_dists, "grad_dists", 4};
  at::CheckedFrom c = "RasterizeSpheresBackwardCuda";
  at::checkAllSameGPU(c, {points_t, idxs_t, grad_zbuf_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, grad_zbuf_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int P = points.size(0);
  const int N = idxs.size(0);
  const int H = idxs.size(1);
  const int W = idxs.size(2);
  const int K = idxs.size(3);

  at::Tensor grad_points = at::zeros({P, 4}, points.options());

  if (grad_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_points;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeSpheresBackwardCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      idxs.contiguous().data_ptr<int32_t>(),
      N, P, H, W, K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_points.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return grad_points;
}
