#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <thrust/tuple.h>
#include <cstdio>
#include <tuple>
#include "rasterize_points/bitmask.cuh"
#include "rasterize_points/rasterization_utils.cuh"
#include "utils/float_math.cuh"
#include "utils/geometry_utils.cuh"

namespace {
// A structure for holding details about a pixel.
struct Pixel {
  float z;
  int64_t idx;
  float dist;
  float3 bary;
};

__device__ bool operator<(const Pixel& a, const Pixel& b) {
  return a.z < b.z;
}

__device__ float FloatMin3(const float p1, const float p2, const float p3) {
  return fminf(p1, fminf(p2, p3));
}

__device__ float FloatMax3(const float p1, const float p2, const float p3) {
  return fmaxf(p1, fmaxf(p2, p3));
}

// Get the xyz coordinates of the three vertices for the face given by the
// index face_idx into face_verts.
__device__ thrust::tuple<float3, float3, float3> GetSingleFaceVerts(
    const float* face_verts,
    int face_idx) {
  const float x0 = face_verts[face_idx * 9 + 0];
  const float y0 = face_verts[face_idx * 9 + 1];
  const float z0 = face_verts[face_idx * 9 + 2];
  const float x1 = face_verts[face_idx * 9 + 3];
  const float y1 = face_verts[face_idx * 9 + 4];
  const float z1 = face_verts[face_idx * 9 + 5];
  const float x2 = face_verts[face_idx * 9 + 6];
  const float y2 = face_verts[face_idx * 9 + 7];
  const float z2 = face_verts[face_idx * 9 + 8];

  const float3 v0xyz = make_float3(x0, y0, z0);
  const float3 v1xyz = make_float3(x1, y1, z1);
  const float3 v2xyz = make_float3(x2, y2, z2);

  return thrust::make_tuple(v0xyz, v1xyz, v2xyz);
}

// Get the min/max x/y/z values for the face given by vertices v0, v1, v2.
__device__ thrust::tuple<float2, float2, float2>
GetFaceBoundingBox(float3 v0, float3 v1, float3 v2) {
  const float xmin = FloatMin3(v0.x, v1.x, v2.x);
  const float ymin = FloatMin3(v0.y, v1.y, v2.y);
  const float zmin = FloatMin3(v0.z, v1.z, v2.z);
  const float xmax = FloatMax3(v0.x, v1.x, v2.x);
  const float ymax = FloatMax3(v0.y, v1.y, v2.y);
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);

  return thrust::make_tuple(
      make_float2(xmin, xmax),
      make_float2(ymin, ymax),
      make_float2(zmin, zmax));
}

// Check if the point (px, py) lies outside the face bounding box face_bbox.
// Return true if the point is outside.
__device__ bool CheckPointOutsideBoundingBox(
    float3 v0,
    float3 v1,
    float3 v2,
    float blur_radius,
    float2 pxy) {
  const auto bbox = GetFaceBoundingBox(v0, v1, v2);
  const float2 xlims = thrust::get<0>(bbox);
  const float2 ylims = thrust::get<1>(bbox);
  const float2 zlims = thrust::get<2>(bbox);

  const float x_min = xlims.x - blur_radius;
  const float y_min = ylims.x - blur_radius;
  const float x_max = xlims.y + blur_radius;
  const float y_max = ylims.y + blur_radius;

  // Faces with at least one vertex behind the camera won't render correctly
  // and should be removed or clipped before calling the rasterizer
  const bool z_invalid = zlims.x < kEpsilon;

  // Check if the current point is oustside the triangle bounding box.
  return (
      pxy.x > x_max || pxy.x < x_min || pxy.y > y_max || pxy.y < y_min ||
      z_invalid);
}

// This function checks if a pixel given by xy location pxy lies within the
// face with index face_idx in face_verts. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the faces which intersect
// with this pixel sorted by closest z distance. If the point pxy lies in the
// face, the list (q) is updated and re-orderered in place. In addition
// the auxillary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizeMeshesNaiveCudaKernel and
// RasterizeMeshesFineCudaKernel.
template <typename FaceQ>
__device__ void CheckPixelInsideFace(
    const float* face_verts, // (F, 3, 3)
    const int face_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    FaceQ& q,
    const float blur_radius,
    const float2 pxy, // Coordinates of the pixel
    const int K,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  const auto v012 = GetSingleFaceVerts(face_verts, face_idx);
  const float3 v0 = thrust::get<0>(v012);
  const float3 v1 = thrust::get<1>(v012);
  const float3 v2 = thrust::get<2>(v012);

  // Only need xy for barycentric coordinates and distance calculations.
  const float2 v0xy = make_float2(v0.x, v0.y);
  const float2 v1xy = make_float2(v1.x, v1.y);
  const float2 v2xy = make_float2(v2.x, v2.y);

  // Perform checks and skip if:
  // 1. the face is behind the camera
  // 2. the face is facing away from the camera
  // 3. the face has very small face area
  // 4. the pixel is outside the face bbox
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);
  const bool outside_bbox = CheckPointOutsideBoundingBox(
      v0, v1, v2, sqrt(blur_radius), pxy); // use sqrt of blur for bbox
  const float face_area = EdgeFunctionForward(v0xy, v1xy, v2xy);
  // Check if the face is visible to the camera.
  const bool back_face = face_area < 0.0;
  const bool zero_face_area = face_area <= kEpsilon && face_area >= -1.0f * kEpsilon;

  if (zmax < 0 || cull_backfaces && back_face || outside_bbox || zero_face_area)
      return;

  // Calculate barycentric coords and euclidean dist to triangle.
  const float3 p_bary0 = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
  const float3 p_bary = !perspective_correct
      ? p_bary0
      : BarycentricPerspectiveCorrectionForward(p_bary0, v0.z, v1.z, v2.z);
  const float3 p_bary_clip =
      !clip_barycentric_coords ? p_bary : BarycentricClipForward(p_bary);

  const float pz =
      p_bary_clip.x * v0.z + p_bary_clip.y * v1.z + p_bary_clip.z * v2.z;

  if (pz < 0)
    return; // Face is behind the image plane.

  // Get abs squared distance
  const float dist = PointTriangleDistanceForward(pxy, v0xy, v1xy, v2xy);

  // Use the unclipped bary coordinates to determine if the point is inside the
  // face.
  const bool inside = p_bary.x > 0.0f && p_bary.y > 0.0f && p_bary.z > 0.0f;
  const float signed_dist = inside ? -dist : dist;

  // Check if pixel is outside blur region
  if (!inside && dist >= blur_radius) {
    return;
  }

  if (q_size < K) {
    // Just insert it.
    q[q_size] = {pz, face_idx, signed_dist, p_bary_clip};
    if (pz > q_max_z) {
      q_max_z = pz;
      q_max_idx = q_size;
    }
    q_size++;
  } else if (pz < q_max_z) {
    // Overwrite the old max, and find the new max.
    q[q_max_idx] = {pz, face_idx, signed_dist, p_bary_clip};
    q_max_z = pz;
    for (int i = 0; i < K; i++) {
      if (q[i].z > q_max_z) {
        q_max_z = q[i].z;
        q_max_idx = i;
      }
    }
  }
}
} // namespace

// ****************************************************************************
// *                          NAIVE RASTERIZATION                      *
// ****************************************************************************
__global__ void RasterizeMeshesNaiveCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const float blur_radius,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces,
    const int N,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs,
    float* zbuf,
    float* pix_dists,
    float* bary) {
  // Simple version: One thread per output pixel
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = tid; i < N * H * W; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (H * W); // batch index.
    const int pix_idx = i % (H * W);

    // Reverse ordering of X and Y axes
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    // screen coordinates to ndc coordiantes of pixel.
    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);
    const float2 pxy = make_float2(xf, yf);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the faces.
    const int64_t face_start_idx = mesh_to_face_first_idx[n];
    const int64_t face_stop_idx = face_start_idx + num_faces_per_mesh[n];

    // Loop through the faces in the mesh.
    for (int f = face_start_idx; f < face_stop_idx; ++f) {
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.

      CheckPixelInsideFace(
          face_verts,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct,
          clip_barycentric_coords,
          cull_backfaces);
    }

    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);
    int idx = n * H * W * K + pix_idx * K;

    for (int k = 0; k < q_size; ++k) {
      face_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist;
      bary[(idx + k) * 3 + 0] = q[k].bary.x;
      bary[(idx + k) * 3 + 1] = q[k].bary.y;
      bary[(idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
RasterizeMeshesNaiveCuda(
    const at::Tensor& face_verts,
    const at::Tensor& mesh_to_faces_packed_first_idx,
    const at::Tensor& num_faces_per_mesh,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int num_closest,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");

  TORCH_CHECK(
      num_faces_per_mesh.size(0) == mesh_to_faces_packed_first_idx.size(0),
      "num_faces_per_mesh must have save size first dimension as mesh_to_faces_packed_first_idx");

  if (num_closest > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      mesh_to_faces_packed_first_idx_t{
          mesh_to_faces_packed_first_idx, "mesh_to_faces_packed_first_idx", 2},
      num_faces_per_mesh_t{num_faces_per_mesh, "num_faces_per_mesh", 3};
  at::CheckedFrom c = "RasterizeMeshesNaiveCuda";
  at::checkAllSameGPU(
      c,
      {face_verts_t, mesh_to_faces_packed_first_idx_t, num_faces_per_mesh_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = num_faces_per_mesh.size(0); // batch size.
  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);
  const int K = num_closest;

  auto long_opts = num_faces_per_mesh.options().dtype(at::kLong);
  auto float_opts = face_verts.options().dtype(at::kFloat);

  at::Tensor face_idxs = at::full({N, H, W, K}, -1, long_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor bary = at::full({N, H, W, K, 3}, -1, float_opts);

  if (face_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesNaiveCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_faces_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      blur_radius,
      perspective_correct,
      clip_barycentric_coords,
      cull_backfaces,
      N,
      H,
      W,
      K,
      face_idxs.data_ptr<int64_t>(),
      zbuf.data_ptr<float>(),
      pix_dists.data_ptr<float>(),
      bary.data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO: benchmark parallelizing over faces_verts instead of over pixels.
__global__ void RasterizeMeshesBackwardCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int64_t* pix_to_face, // (N, H, W, K)
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const int N,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_bary, // (N, H, W, K, 3)
    const float* grad_dists, // (N, H, W, K)
    float* grad_face_verts) { // (F, 3, 3)

  // Parallelize over each pixel in images of
  // size H * W, for each image in the batch of size N.
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int t_i = tid; t_i < N * H * W; t_i += num_threads) {
    // Convert linear index to 3D index
    const int n = t_i / (H * W); // batch index.
    const int pix_idx = t_i % (H * W);

    // Reverse ordering of X and Y axes.
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);
    const float2 pxy = make_float2(xf, yf);

    // Loop over all the faces for this pixel.
    for (int k = 0; k < K; k++) {
      // Index into (N, H, W, K, :) grad tensors
      // pixel index + top k index
      int i = n * H * W * K + pix_idx * K + k;

      const int f = pix_to_face[i];
      if (f < 0) {
        continue; // padded face.
      }
      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Only neex xy for barycentric coordinate and distance calculations.
      const float2 v0xy = make_float2(v0.x, v0.y);
      const float2 v1xy = make_float2(v1.x, v1.y);
      const float2 v2xy = make_float2(v2.x, v2.y);

      // Get upstream gradients for the face.
      const float grad_dist_upstream = grad_dists[i];
      const float grad_zbuf_upstream = grad_zbuf[i];
      const float grad_bary_upstream_w0 = grad_bary[i * 3 + 0];
      const float grad_bary_upstream_w1 = grad_bary[i * 3 + 1];
      const float grad_bary_upstream_w2 = grad_bary[i * 3 + 2];
      const float3 grad_bary_upstream = make_float3(
          grad_bary_upstream_w0, grad_bary_upstream_w1, grad_bary_upstream_w2);

      const float3 b_w = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
      const float3 b_pp = !perspective_correct
          ? b_w
          : BarycentricPerspectiveCorrectionForward(b_w, v0.z, v1.z, v2.z);

      const float3 b_w_clip =
          !clip_barycentric_coords ? b_pp : BarycentricClipForward(b_pp);

      const bool inside = b_pp.x > 0.0f && b_pp.y > 0.0f && b_pp.z > 0.0f;
      const float sign = inside ? -1.0f : 1.0f;

      auto grad_dist_f = PointTriangleDistanceBackward(
          pxy, v0xy, v1xy, v2xy, sign * grad_dist_upstream);
      const float2 ddist_d_v0 = thrust::get<1>(grad_dist_f);
      const float2 ddist_d_v1 = thrust::get<2>(grad_dist_f);
      const float2 ddist_d_v2 = thrust::get<3>(grad_dist_f);

      // Upstream gradient for barycentric coords from zbuf calculation:
      // zbuf = bary_w0 * z0 + bary_w1 * z1 + bary_w2 * z2
      // Therefore
      // d_zbuf/d_bary_w0 = z0
      // d_zbuf/d_bary_w1 = z1
      // d_zbuf/d_bary_w2 = z2
      const float3 d_zbuf_d_bwclip = make_float3(v0.z, v1.z, v2.z);

      // Total upstream barycentric gradients are the sum of
      // external upstream gradients and contribution from zbuf.
      const float3 grad_bary_f_sum =
          (grad_bary_upstream + grad_zbuf_upstream * d_zbuf_d_bwclip);

      float3 grad_bary0 = grad_bary_f_sum;

      if (clip_barycentric_coords) {
        grad_bary0 = BarycentricClipBackward(b_w, grad_bary_f_sum);
      }

      float dz0_persp = 0.0f, dz1_persp = 0.0f, dz2_persp = 0.0f;
      if (perspective_correct) {
        auto perspective_grads = BarycentricPerspectiveCorrectionBackward(
            b_w, v0.z, v1.z, v2.z, grad_bary0);
        grad_bary0 = thrust::get<0>(perspective_grads);
        dz0_persp = thrust::get<1>(perspective_grads);
        dz1_persp = thrust::get<2>(perspective_grads);
        dz2_persp = thrust::get<3>(perspective_grads);
      }

      auto grad_bary_f =
          BarycentricCoordsBackward(pxy, v0xy, v1xy, v2xy, grad_bary0);
      const float2 dbary_d_v0 = thrust::get<1>(grad_bary_f);
      const float2 dbary_d_v1 = thrust::get<2>(grad_bary_f);
      const float2 dbary_d_v2 = thrust::get<3>(grad_bary_f);

      atomicAdd(grad_face_verts + f * 9 + 0, dbary_d_v0.x + ddist_d_v0.x);
      atomicAdd(grad_face_verts + f * 9 + 1, dbary_d_v0.y + ddist_d_v0.y);
      atomicAdd(
          grad_face_verts + f * 9 + 2,
          grad_zbuf_upstream * b_w_clip.x + dz0_persp);
      atomicAdd(grad_face_verts + f * 9 + 3, dbary_d_v1.x + ddist_d_v1.x);
      atomicAdd(grad_face_verts + f * 9 + 4, dbary_d_v1.y + ddist_d_v1.y);
      atomicAdd(
          grad_face_verts + f * 9 + 5,
          grad_zbuf_upstream * b_w_clip.y + dz1_persp);
      atomicAdd(grad_face_verts + f * 9 + 6, dbary_d_v2.x + ddist_d_v2.x);
      atomicAdd(grad_face_verts + f * 9 + 7, dbary_d_v2.y + ddist_d_v2.y);
      atomicAdd(
          grad_face_verts + f * 9 + 8,
          grad_zbuf_upstream * b_w_clip.z + dz2_persp);
    }
  }
}

at::Tensor RasterizeMeshesBackwardCuda(
    const at::Tensor& face_verts, // (F, 3, 3)
    const at::Tensor& pix_to_face, // (N, H, W, K)
    const at::Tensor& grad_zbuf, // (N, H, W, K)
    const at::Tensor& grad_bary, // (N, H, W, K, 3)
    const at::Tensor& grad_dists, // (N, H, W, K)
    const bool perspective_correct,
    const bool clip_barycentric_coords) {
  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      pix_to_face_t{pix_to_face, "pix_to_face", 2},
      grad_zbuf_t{grad_zbuf, "grad_zbuf", 3},
      grad_bary_t{grad_bary, "grad_bary", 4},
      grad_dists_t{grad_dists, "grad_dists", 5};
  at::CheckedFrom c = "RasterizeMeshesBackwardCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, pix_to_face_t, grad_zbuf_t, grad_bary_t, grad_dists_t});
  at::checkAllSameType(
      c, {face_verts_t, grad_zbuf_t, grad_bary_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int F = face_verts.size(0);
  const int N = pix_to_face.size(0);
  const int H = pix_to_face.size(1);
  const int W = pix_to_face.size(2);
  const int K = pix_to_face.size(3);

  at::Tensor grad_face_verts = at::zeros({F, 3, 3}, face_verts.options());

  if (grad_face_verts.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_face_verts;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesBackwardCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      pix_to_face.contiguous().data_ptr<int64_t>(),
      perspective_correct,
      clip_barycentric_coords,
      N,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_bary.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_face_verts.data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return grad_face_verts;
}

// ****************************************************************************
// *                          COARSE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizeMeshesCoarseCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const float blur_radius,
    const int N,
    const int F,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_faces_per_bin,
    int* faces_per_bin,
    int* bin_faces) {
  extern __shared__ char sbuf[];
  const int M = max_faces_per_bin;
  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of faces
  const int chunks_per_batch = 1 + (F - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;

  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch; // batch index
    const int chunk_idx = chunk % chunks_per_batch;
    const int face_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();
    const int64_t mesh_face_start_idx = mesh_to_face_first_idx[batch_idx];
    const int64_t mesh_face_stop_idx =
        mesh_face_start_idx + num_faces_per_mesh[batch_idx];

    // Have each thread handle a different face within the chunk
    for (int f = threadIdx.x; f < chunk_size; f += blockDim.x) {
      const int f_idx = face_start_idx + f;

      // Check if face index corresponds to the mesh in the batch given by
      // batch_idx
      if (f_idx >= mesh_face_stop_idx || f_idx < mesh_face_start_idx) {
        continue;
      }

      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f_idx);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Compute screen-space bbox for the triangle expanded by blur.
      float xmin = FloatMin3(v0.x, v1.x, v2.x) - sqrt(blur_radius);
      float ymin = FloatMin3(v0.y, v1.y, v2.y) - sqrt(blur_radius);
      float xmax = FloatMax3(v0.x, v1.x, v2.x) + sqrt(blur_radius);
      float ymax = FloatMax3(v0.y, v1.y, v2.y) + sqrt(blur_radius);
      float zmin = FloatMin3(v0.z, v1.z, v2.z);

      // Faces with at least one vertex behind the camera won't render
      // correctly and should be removed or clipped before calling the
      // rasterizer
      if (zmin < kEpsilon) {
        continue;
      }

      // Brute-force search over all bins; TODO(T54294966) something smarter.
      for (int by = 0; by < num_bins_y; ++by) {
        // Y coordinate of the top and bottom of the bin.
        // PixToNdc gives the location of the center of each pixel, so we
        // need to add/subtract a half pixel to get the true extent of the bin.
        // Reverse ordering of Y axis so that +Y is upwards in the image.
        const float bin_y_min =
            PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float bin_y_max =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (ymin <= bin_y_max) && (bin_y_min < ymax);

        for (int bx = 0; bx < num_bins_x; ++bx) {
          // X coordinate of the left and right of the bin.
          // Reverse ordering of x axis so that +X is left.
          const float bin_x_max =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const float bin_x_min =
              PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;

          const bool x_overlap = (xmin <= bin_x_max) && (bin_x_min < xmax);
          if (y_overlap && x_overlap) {
            binmask.set(by, bx, f);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every face in the current chunk. We need to
    // count the number of faces in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int faces_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of faces found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_faces array for the
      // faces in the current chunk that fall into this bin.
      const int start = atomicAdd(faces_per_bin + faces_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_faces.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int f = 0; f < chunk_size; ++f) {
        if (binmask.get(by, bx, f)) {
          // TODO(T54296346) find the correct method for handling errors in
          // CUDA. Throw an error if num_faces_per_bin > max_faces_per_bin.
          // Either decrease bin size or increase max_faces_per_bin
          bin_faces[next_idx] = face_start_idx + f;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizeMeshesCoarseCuda(
    const at::Tensor& face_verts,
    const at::Tensor& mesh_to_face_first_idx,
    const at::Tensor& num_faces_per_mesh,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int bin_size,
    const int max_faces_per_bin) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      mesh_to_face_first_idx_t{
          mesh_to_face_first_idx, "mesh_to_face_first_idx", 2},
      num_faces_per_mesh_t{num_faces_per_mesh, "num_faces_per_mesh", 3};
  at::CheckedFrom c = "RasterizeMeshesCoarseCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, mesh_to_face_first_idx_t, num_faces_per_mesh_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int F = face_verts.size(0);
  const int N = num_faces_per_mesh.size(0);
  const int M = max_faces_per_bin;

  // Integer divide round up.
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    std::stringstream ss;
    ss << "In Coarse Rasterizer got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_faces_per_mesh.options().dtype(at::kInt);
  at::Tensor faces_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_faces = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_faces.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_faces;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizeMeshesCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_face_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      blur_radius,
      N,
      F,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      faces_per_bin.data_ptr<int32_t>(),
      bin_faces.data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_faces;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************
__global__ void RasterizeMeshesFineCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int32_t* bin_faces, // (N, BH, BW, T)
    const float blur_radius,
    const int bin_size,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces,
    const int N,
    const int BH,
    const int BW,
    const int M,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists, // (N, H, W, K)
    float* bary // (N, H, W, K, 3)
) {
  // This can be more than H * W if H or W are not divisible by bin_size.
  int num_pixels = N * BH * BW * bin_size * bin_size;
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from faces and bin_faces.
    int i = pid;
    const int n = i / (BH * BW * bin_size * bin_size);
    i %= BH * BW * bin_size * bin_size;
    // bin index y
    const int by = i / (BW * bin_size * bin_size);
    i %= BW * bin_size * bin_size;
    // bin index y
    const int bx = i / (bin_size * bin_size);
    // pixel within the bin
    i %= bin_size * bin_size;

    // Pixel x, y indices
    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= H || xi >= W)
      continue;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    const float2 pxy = make_float2(xf, yf);

    // This part looks like the naive rasterization kernel, except we use
    // bin_faces to only look at a subset of faces already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; m++) {
      const int f = bin_faces[n * BH * BW * M + by * BW * M + bx * M + m];
      if (f < 0) {
        continue; // bin_faces uses -1 as a sentinal value.
      }
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.
      CheckPixelInsideFace(
          face_verts,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct,
          clip_barycentric_coords,
          cull_backfaces);
    }

    // Now we've looked at all the faces for this bin, so we can write
    // output for the current pixel.
    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis so that
    // in the image +Y is pointing up and +X is pointing left.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const int pix_idx = n * H * W * K + yidx * W * K + xidx * K;
    for (int k = 0; k < q_size; k++) {
      face_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist;
      bary[(pix_idx + k) * 3 + 0] = q[k].bary.x;
      bary[(pix_idx + k) * 3 + 1] = q[k].bary.y;
      bary[(pix_idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
RasterizeMeshesFineCuda(
    const at::Tensor& face_verts,
    const at::Tensor& bin_faces,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int bin_size,
    const int faces_per_pixel,
    const bool perspective_correct,
    const bool clip_barycentric_coords,
    const bool cull_backfaces) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");
  TORCH_CHECK(bin_faces.ndimension() == 4, "bin_faces must have 4 dimensions");

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      bin_faces_t{bin_faces, "bin_faces", 2};
  at::CheckedFrom c = "RasterizeMeshesFineCuda";
  at::checkAllSameGPU(c, {face_verts_t, bin_faces_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // bin_faces shape (N, BH, BW, M)
  const int N = bin_faces.size(0);
  const int BH = bin_faces.size(1);
  const int BW = bin_faces.size(2);
  const int M = bin_faces.size(3);
  const int K = faces_per_pixel;

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 150");
  }
  auto long_opts = bin_faces.options().dtype(at::kLong);
  auto float_opts = face_verts.options().dtype(at::kFloat);

  at::Tensor face_idxs = at::full({N, H, W, K}, -1, long_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor bary = at::full({N, H, W, K, 3}, -1, float_opts);

  if (face_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesFineCudaKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      bin_faces.contiguous().data_ptr<int32_t>(),
      blur_radius,
      bin_size,
      perspective_correct,
      clip_barycentric_coords,
      cull_backfaces,
      N,
      BH,
      BW,
      M,
      H,
      W,
      K,
      face_idxs.data_ptr<int64_t>(),
      zbuf.data_ptr<float>(),
      pix_dists.data_ptr<float>(),
      bary.data_ptr<float>());

  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}
